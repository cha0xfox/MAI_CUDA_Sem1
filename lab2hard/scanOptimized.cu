#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <vector> // Для векторов
#include <algorithm> // Для generate

#include <hip/hip_runtime.h> 

// Для работы curand
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

using std::vector;
using std::generate;
using std::cout;
using std::cerr;
using std::endl;

void scanLargeDeviceArray(int *output, int *input, int length, bool bcao);
void scanSmallDeviceArray(int *d_out, int *d_in, int length, bool bcao);
void scanLargeEvenDeviceArray(int *output, int *input, int length, bool bcao);

__global__ void prescan_arbitrary(int *output, int *input, int n, int powerOfTwo);
__global__ void prescan_arbitrary_unoptimized(int *output, int *input, int n, int powerOfTwo);

__global__ void prescan_large(int *output, int *input, int n, int* sums);
__global__ void prescan_large_unoptimized(int *output, int *input, int n, int *sums);

__global__ void add(int *output, int length, int *n1);
__global__ void add(int *output, int length, int *n1, int *n2);

bool isPowerOfTwo(int x);
int nextPowerOfTwo(int x);

int THREADS_PER_BLOCK = 512;
int ELEMENTS_PER_BLOCK = THREADS_PER_BLOCK * 2;

#define SHARED_MEMORY_BANKS 32
#define LOG_MEM_BANKS 5
// Оптимизация конфликта банков
#define CONFLICT_FREE_OFFSET(n) ((n) >> LOG_MEM_BANKS)

// Конструкция проверок на ошибки
#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err, const char* const func, const char* const file,
  const int line)
  {
    if (err != hipSuccess)
    {
      std::cerr << "CUDA Runtime Error at: " << file << ":" << line
      << std::endl;
      std::cerr << hipGetErrorString(err) << " " << func << std::endl;
    }
  }
  
#define CHECK_LAST_CUDA_ERROR() checkLast(__FILE__, __LINE__)
void checkLast(const char* const file, const int line)
{
    hipError_t err{hipGetLastError()};
    if (err != hipSuccess)
    {
      std::cerr << "CUDA Runtime Error at: " << file << ":" << line
      << std::endl;
      std::cerr << hipGetErrorString(err) << std::endl;
    }
}
// ============================ LOGIC FUNCS

void scanLargeDeviceArray(int *d_out, int *d_in, int length, bool bcao) {
	int remainder = length % (ELEMENTS_PER_BLOCK);
	if (remainder == 0) {
		scanLargeEvenDeviceArray(d_out, d_in, length, bcao);
	}
	else {
		// Делаем скан по всему массиву
		int lengthMultiple = length - remainder;
		scanLargeEvenDeviceArray(d_out, d_in, lengthMultiple, bcao);

		// Скан по оставшемуся массиву (более меньшему), сохраняя последнее число.
		int *startOfOutputArray = &(d_out[lengthMultiple]);
		scanSmallDeviceArray(startOfOutputArray, &(d_in[lengthMultiple]), remainder, bcao);

		add<<<1, remainder>>>(startOfOutputArray, remainder, &(d_in[lengthMultiple - 1]), &(d_out[lengthMultiple - 1]));
	}
}

void scanSmallDeviceArray(int *d_out, int *d_in, int length, bool bcao) {
	int powerOfTwo = nextPowerOfTwo(length);

	if (bcao) {
		prescan_arbitrary<<<1, (length + 1) / 2, 2 * powerOfTwo * sizeof(int)>>>(d_out, d_in, length, powerOfTwo);
	}
	else {
		prescan_arbitrary_unoptimized<<<1, (length + 1) / 2, 2 * powerOfTwo * sizeof(int)>>>(d_out, d_in, length, powerOfTwo);
	}
}

void scanLargeEvenDeviceArray(int *d_out, int *d_in, int length, bool bcao) {
	const int blocks = length / ELEMENTS_PER_BLOCK;
	const int sharedMemArraySize = ELEMENTS_PER_BLOCK * sizeof(int);

	int *d_sums, *d_incr;
	hipMalloc((void **)&d_sums, blocks * sizeof(int));
	hipMalloc((void **)&d_incr, blocks * sizeof(int));

	if (bcao) {
		prescan_large<<<blocks, THREADS_PER_BLOCK, 2 * sharedMemArraySize>>>(d_out, d_in, ELEMENTS_PER_BLOCK, d_sums);
	}
	else {
		prescan_large_unoptimized<<<blocks, THREADS_PER_BLOCK, 2 * sharedMemArraySize>>>(d_out, d_in, ELEMENTS_PER_BLOCK, d_sums);
	}

	const int sumsArrThreadsNeeded = (blocks + 1) / 2;
	if (sumsArrThreadsNeeded > THREADS_PER_BLOCK) {
		// Делаем скан по всему массиву
		scanLargeDeviceArray(d_incr, d_sums, blocks, bcao);
	}
	else {
		// Нужен только один блок на скан всего массива
		scanSmallDeviceArray(d_incr, d_sums, blocks, bcao);
	}

	add<<<blocks, ELEMENTS_PER_BLOCK>>>(d_out, ELEMENTS_PER_BLOCK, d_incr);

	hipFree(d_sums);
	hipFree(d_incr);
}

// ============================ KERNEL FUNCS

__global__ void prescan_arbitrary(int *output, int *input, int n, int powerOfTwo){
	extern __shared__ int temp[];// allocated on invocation
	int threadID = threadIdx.x;

	int ai = threadID;
	int bi = threadID + (n / 2);
	int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
	int bankOffsetB = CONFLICT_FREE_OFFSET(bi);


	if (threadID < n) {
		temp[ai + bankOffsetA] = input[ai];
		temp[bi + bankOffsetB] = input[bi];
	}
	else {
		temp[ai + bankOffsetA] = 0;
		temp[bi + bankOffsetB] = 0;
	}


	int offset = 1;
	for (int d = powerOfTwo >> 1; d > 0; d >>= 1) // строим дерево
	{
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);

			temp[bi] += temp[ai];
		}
		offset *= 2;
	}

	if (threadID == 0) {
		temp[powerOfTwo - 1 + CONFLICT_FREE_OFFSET(powerOfTwo - 1)] = 0; // очищаем последний элемент
	}

	for (int d = 1; d < powerOfTwo; d *= 2) // Проходим по дереву вниз
	{
		offset >>= 1;
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);

			int t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();

	if (threadID < n) {
		output[ai] = temp[ai + bankOffsetA];
		output[bi] = temp[bi + bankOffsetB];
	}
}

__global__ void prescan_arbitrary_unoptimized(int *output, int *input, int n, int powerOfTwo) {
	extern __shared__ int temp[];// allocated on invocation
	int threadID = threadIdx.x;

	if (threadID < n) {
		temp[2 * threadID] = input[2 * threadID]; // загружаем массив в память
		temp[2 * threadID + 1] = input[2 * threadID + 1];
	}
	else {
		temp[2 * threadID] = 0;
		temp[2 * threadID + 1] = 0;
	}


	int offset = 1;
	for (int d = powerOfTwo >> 1; d > 0; d >>= 1) // строим дерево
	{
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			temp[bi] += temp[ai];
		}
		offset *= 2;
	}

	if (threadID == 0) { temp[powerOfTwo - 1] = 0; } // очищаем последний элемент

	for (int d = 1; d < powerOfTwo; d *= 2) // Проходим по дереву вниз
	{
		offset >>= 1;
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			int t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();

	if (threadID < n) {
		output[2 * threadID] = temp[2 * threadID]; // Записываем в память устройства
		output[2 * threadID + 1] = temp[2 * threadID + 1];
	}
}


__global__ void prescan_large(int *output, int *input, int n, int *sums) {
	extern __shared__ int temp[];

	int blockID = blockIdx.x;
	int threadID = threadIdx.x;
	int blockOffset = blockID * n;

	int ai = threadID;
	int bi = threadID + (n / 2);
	int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
	int bankOffsetB = CONFLICT_FREE_OFFSET(bi);
	temp[ai + bankOffsetA] = input[blockOffset + ai];
	temp[bi + bankOffsetB] = input[blockOffset + bi];

	int offset = 1;
	for (int d = n >> 1; d > 0; d >>= 1) // строим дерево
	{
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);

			temp[bi] += temp[ai];
		}
		offset *= 2;
	}
	__syncthreads();


	if (threadID == 0) {
		sums[blockID] = temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)];
		temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)] = 0;
	}

	for (int d = 1; d < n; d *= 2) // Проходим по дереву вниз
	{
		offset >>= 1;
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);

			int t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();

	output[blockOffset + ai] = temp[ai + bankOffsetA];
	output[blockOffset + bi] = temp[bi + bankOffsetB];
}

__global__ void prescan_large_unoptimized(int *output, int *input, int n, int *sums) {
	int blockID = blockIdx.x;
	int threadID = threadIdx.x;
	int blockOffset = blockID * n;

	extern __shared__ int temp[];
	temp[2 * threadID] = input[blockOffset + (2 * threadID)];
	temp[2 * threadID + 1] = input[blockOffset + (2 * threadID) + 1];

	int offset = 1;
	for (int d = n >> 1; d > 0; d >>= 1) // строим дерево
	{
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			temp[bi] += temp[ai];
		}
		offset *= 2;
	}
	__syncthreads();


	if (threadID == 0) {
		sums[blockID] = temp[n - 1];
		temp[n - 1] = 0;
	}

	for (int d = 1; d < n; d *= 2) // Проходим по дереву вниз
	{
		offset >>= 1;
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			int t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();

	output[blockOffset + (2 * threadID)] = temp[2 * threadID];
	output[blockOffset + (2 * threadID) + 1] = temp[2 * threadID + 1];
}


__global__ void add(int *output, int length, int *n) {
	int blockID = blockIdx.x;
	int threadID = threadIdx.x;
	int blockOffset = blockID * length;

	output[blockOffset + threadID] += n[blockID];
}

__global__ void add(int *output, int length, int *n1, int *n2) {
	int blockID = blockIdx.x;
	int threadID = threadIdx.x;
	int blockOffset = blockID * length;

	output[blockOffset + threadID] += n1[blockID] + n2[blockID];
}

// ============================
// Является ли степенью двойки
bool isPowerOfTwo(int x) {
	return x && !(x & (x - 1));
}

// Следующая степень двойки
int nextPowerOfTwo(int x) {
	int power = 1;
	while (power < x) {
		power *= 2;
	}
	return power;
}

// ============================
// C++11 way генерация случайных чисел с нормальным распределением

void testCPU(int* output, int* input, int length) {
	output[0] = 0; // первый элемент 0, так как это по сути прескан
	for (int j = 1; j < length; ++j)
	{
		output[j] = input[j - 1] + output[j - 1];
	}
}

int main() {

  using clock = std::chrono::system_clock;
  using sec = std::chrono::duration<double, std::milli>;

  constexpr int N = 10000000;

  int length = N;
  bool bcao = false;

  vector<int> h_a(N);
  vector<int> h_b(N);
  generate(h_a.begin(), h_a.end(), []() { return rand() % 10; });

  int *d_a, *d_b;

  CHECK_CUDA_ERROR(hipMalloc(&d_a, N * sizeof(int)));
  CHECK_CUDA_ERROR(hipMalloc(&d_b, N * sizeof(int)));
  CHECK_CUDA_ERROR(hipMemcpy(d_a, h_a.data(), N * sizeof(int), hipMemcpyHostToDevice));

  // Создаем эвенты, которые рассчитают время выполнения на GPU.
  hipEvent_t start,stop;
  float gpuTime = 0.0f;
  CHECK_CUDA_ERROR(hipEventCreate(&start));
  CHECK_CUDA_ERROR(hipEventCreate(&stop));

  CHECK_CUDA_ERROR(hipEventRecord(start,0));

  // Запускаем в работу GPU асинхронно
  if (length > ELEMENTS_PER_BLOCK) {
		scanLargeDeviceArray(d_b, d_a, length, bcao);
	}
	else {
		scanSmallDeviceArray(d_b, d_a, length, bcao);
	}

  CHECK_CUDA_ERROR(hipEventRecord(stop,0));
  CHECK_CUDA_ERROR(hipEventSynchronize(stop));
  CHECK_CUDA_ERROR(hipEventElapsedTime(&gpuTime, start, stop));
  CHECK_CUDA_ERROR(hipMemcpy(h_b.data(), d_b, N*sizeof(int), hipMemcpyDeviceToHost));

  // Уничтожаем созданные эвенты
  CHECK_CUDA_ERROR(hipEventDestroy(start));
  CHECK_CUDA_ERROR(hipEventDestroy(stop));

  CHECK_LAST_CUDA_ERROR();
  // ----------------------------
  
  cout << "GPU array: ";
  for (int i = N; i > N - 10; i--){
    cout << "[" << h_b[i] << "] ";
  }
  cout << endl;
  
  // Тестируем время выполнения на CPU
  const auto before = clock::now();

  testCPU(h_b.data(),h_a.data(),N);

  const sec duration = clock::now() - before;

  
  cout << "CPU array: ";
  for (int i = N; i > N - 10; i--){
    cout << "[" << h_b[i] << "] ";
  }
  cout << endl;
  
  // Вывод времени
  cout << "Time elapsed on GPU: " << gpuTime << "ms" << endl;
  cout << "Time elapsed on CPU: " << duration.count() << "ms" << endl;
  
  CHECK_CUDA_ERROR(hipFree(d_a));
  CHECK_CUDA_ERROR(hipFree(d_b));

  return 0;
} 